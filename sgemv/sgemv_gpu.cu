#include <stdio.h>
#include "hipblas.h"
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#define BLOCK_SIZE 16 // Threads per block supported by the GPU


__global__ void sgemv_kernel (char Trans, int M, int N, float Alpha, float *A, float * X, float * Y , float Beta) {
	int y_size = (Trans == 'n' || Trans == 'N')? M:N;
	int x_size = (Trans == 'n' || Trans == 'N')? N:M;
	int elementId = blockIdx.x * BLOCK_SIZE + threadIdx.x; 

	if(elementId<y_size) {
		
		float sum = 0;
		for (int i = 0 ;i <x_size;i++) {
			sum += A[elementId *x_size  + i]   * X[i]; 
		}
		sum = sum * Alpha;
		
		Y[elementId] = Y[elementId] * Beta + sum;

	}

}

int Ceil (int M , int x) {
	if(M%x ==0) {
		return M/x;
	} else {
		return M/x+1;
	}
}


void sgemv(char Trans, int M, int N, float Alpha,float* A, float * X, float*Y , float Beta) {
	float * d_A, *d_X, *d_Y;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);

	hipMalloc((void**) &d_A, sizeof(float) * M*N);
	hipMemcpy(d_A, A , sizeof(float) * M*N , hipMemcpyHostToDevice);

	if(Trans=='n' || Trans == 'N') {
		hipMalloc((void**) &d_X, sizeof(float) *N);
		hipMemcpy(d_X, X , sizeof(float) *N , hipMemcpyHostToDevice);
		
		hipMalloc((void**) &d_Y, sizeof(float) * M);
		hipMemcpy(d_Y, Y , sizeof(float) * M , hipMemcpyHostToDevice);
	} else {
		hipMalloc((void**) &d_X, sizeof(float) * M);
		hipMemcpy(d_X, X , sizeof(float) * M , hipMemcpyHostToDevice);
		
		hipMalloc((void**) &d_Y, sizeof(float) * N);
		hipMemcpy(d_Y, Y , sizeof(float) *N , hipMemcpyHostToDevice);
	}
	int y_size = (Trans == 'n' || Trans == 'N')? M:N;


	/*Determine Kernel Parameters*/
	int num_blocks = Ceil(y_size, BLOCK_SIZE);
	 

	/*Launch Kernel*/
	sgemv_kernel <<<num_blocks, BLOCK_SIZE >>> (Trans, M, N, Alpha , d_A, d_X, d_Y, Beta);


	/* Copy Memory Back to Host */
	if (Trans == 'n' || Trans == 'N') {
		hipMemcpy(Y, d_Y, sizeof(float) * M , hipMemcpyDeviceToHost);
	} else {
		hipMemcpy(Y, d_Y, sizeof(float) * N, hipMemcpyDeviceToHost);
	}


	/* Free Device Memory*/
	hipFree(d_A);
	hipFree(d_X);
	hipFree(d_Y);
	
	hipEventRecord(stop, 0);
	float timeElapsed;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timeElapsed, start, stop);
	printf("Time Elapsed : %f\n", timeElapsed);
}



int main(int argc, char * argv[]) {
	char Trans; 
	int M, N;
	float Alpha; float Beta;
	float* A; // Matrix A
	float * X;
	float * Y;
	int INCY = 1;


	/* 
		Test Cases are to be input
	*/
	
	scanf("%c %d %d %f %f",&Trans, &M, &N, &Alpha, &Beta );
	
	hipHostMalloc((void**) &A, sizeof(float) * M*N, hipHostMallocDefault);
	int y_size, x_size; 
	
	if (Trans == 'N' || Trans == 'n') {
		hipHostMalloc((void**) &X, sizeof(float)*N, hipHostMallocDefault);
		hipHostMalloc((void**) &Y, sizeof(float)*M, hipHostMallocDefault);
		y_size = M;
		x_size = N;
	} else {
		hipHostMalloc((void**) &X, sizeof(float)*M, hipHostMallocDefault);
		hipHostMalloc((void**) &Y, sizeof(float)*N, hipHostMallocDefault);
		y_size = N;
		x_size = M;
	}
	
	//	printf("Y_Size %c %d %d %f %f\n", Trans, y_size, x_size, Alpha, Beta);
	for (int i= 0; i<M;i++) {
		for (int j = 0; j<N;j++) {
			scanf("%f", A + i*N + j);
		}
	}

	for (int i= 0; i<x_size;i++) {
		scanf("%f", X + i);
	}

	for (int i= 0; i<y_size;i++) {
		scanf("%f", Y + i);
	}


	sgemv(Trans, M, N , Alpha, A,X,Y,5);
	/* Display Output */
	FILE * fp;
	fp = fopen("Results/sgemv_gpu_last.txt", "w");
	for(int i =0; i<y_size;i++) {
		fprintf(fp,"%f ", Y[i]);
	}
	fclose(fp);

}
