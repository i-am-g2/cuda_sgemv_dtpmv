#include <stdio.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#include <hip/hip_runtime.h>

#include "hipblas.h"

/* Cublas Works in Column Major and 1 based indexing :: TAKE CARE OF THIS*/

void sgemv(char Trans, int M, int N, float Alpha, float* A, float * X, float*Y , float Beta) {
	float * d_A, *d_X, *d_Y;

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);

	hipMalloc((void**) &d_A, sizeof(float) * M*N);
	hipMemcpy(d_A, A , sizeof(float) * M*N , hipMemcpyHostToDevice);

	if(Trans=='n' || Trans == 'N') {
		hipMalloc((void**) &d_X, sizeof(float) *N);
		hipMemcpy(d_X, X , sizeof(float) *N , hipMemcpyHostToDevice);
		
		hipMalloc((void**) &d_Y, sizeof(float) * M);
		hipMemcpy(d_Y, Y , sizeof(float) * M , hipMemcpyHostToDevice);
	} else {
		hipMalloc((void**) &d_X, sizeof(float) * M);
		hipMemcpy(d_X, X , sizeof(float) * M , hipMemcpyHostToDevice);
		
		hipMalloc((void**) &d_Y, sizeof(float) * N);
		hipMemcpy(d_Y, Y , sizeof(float) *N , hipMemcpyHostToDevice);
	}
	

	if (Trans == 'N') {
		hipblasSgemv(
					handle, HIPBLAS_OP_N,  
					M, N, 
					&Alpha, 
					d_A, M, 
					d_X, 1,
					&Beta, 
					d_Y, 1 
				);
	} else {
		hipblasSgemv(handle, HIPBLAS_OP_T,  M, N, &Alpha,  d_A, M, d_X, 1,&Beta, d_Y, 1 );
	}


	/* Copy Memory Back to Host */
	if (Trans == 'n' || Trans == 'N') {
		hipMemcpy(Y, d_Y, sizeof(float) * M , hipMemcpyDeviceToHost);
	} else {
		hipMemcpy(Y, d_Y, sizeof(float) * N, hipMemcpyDeviceToHost);
	}


	/* Free Device Memory*/
	hipFree(d_A);
	hipFree(d_X);
	hipFree(d_Y);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);	
	float timeElapsed;
	hipEventElapsedTime (&timeElapsed, start, stop);
	printf("Time Elapsed : %f ms\n", timeElapsed);
}



int main(int argc , char * argv[]) {
	char Trans; 
	int M, N;
	float Alpha; float Beta;
	float* A; // Matrix A
	float * X;
	float * Y;
	const int INCY = 1;

	/* 
		Test Cases are to be input
	*/
	
	scanf("%c %d %d %f %f",&Trans, &M, &N, &Alpha, &Beta );

	hipHostMalloc((void**) &A, sizeof(float) * M*N, hipHostMallocDefault);
	int y_size, x_size; 
	
	if (Trans == 'N' || Trans == 'n') {
		hipHostMalloc((void**) &X, sizeof(float)*N, hipHostMallocDefault);
		hipHostMalloc((void**) &Y, sizeof(float)*M, hipHostMallocDefault);
		y_size = M;
		x_size = N;
	} else {
		hipHostMalloc((void**) &X, sizeof(float)*M, hipHostMallocDefault);
		hipHostMalloc((void**) &Y, sizeof(float)*N, hipHostMallocDefault);
		y_size = N;
		x_size = M;
	}
	

	for (int i= 0; i<M;i++) {
		for (int j = 0; j<N;j++) {
			scanf("%f", A + j*M + i);
		}
	}

	for (int i= 0; i<x_size;i++) {
		scanf("%f", X + i);
	}

	for (int i= 0; i<y_size;i++) {
		scanf("%f", Y + i);
	}

	/* Cublas Call */
	


	sgemv(Trans, M, N , Alpha, A,X,Y,Beta);
	/* Display Output */
	FILE * fp;
	fp = fopen("Results/sgemv_cublas_last.txt", "w");
	for(int i =0; i<y_size;i++) {
		fprintf(fp,"%lf ", Y[i]);
	}
	fclose(fp);

}
