#include <stdio.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"
#define BLOCK_SIZE 16 // Threads per block supported by the GPU

void dtpmv(char UPLO, char TRANS, char DIAG, int N, double * A, double * X) {

	double * d_A, *d_X;
	int size_a = (N*(N+1))/2;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);

	hipMalloc((void**) &d_A, sizeof(double)*size_a );
	hipMalloc((void**) &d_X, sizeof(double)*N );

	hipMemcpy(d_A, A , sizeof(double) *size_a, hipMemcpyHostToDevice);
	hipMemcpy(d_X, X, sizeof(double) * N, hipMemcpyHostToDevice);
	
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	auto cudaUPLOMode = HIPBLAS_FILL_MODE_UPPER;
	auto transMode = HIPBLAS_OP_N;
	if (UPLO == 'L') {
		cudaUPLOMode = HIPBLAS_FILL_MODE_LOWER;
	}
	if (TRANS=='T') {
		transMode = HIPBLAS_OP_T;
	}

	hipblasDtpmv(handle, cudaUPLOMode, transMode, HIPBLAS_DIAG_NON_UNIT,N, d_A, d_X, 1);

	/* Copy Memory Back to Host */
	hipMemcpy(X, d_X, sizeof(double) * N , hipMemcpyDeviceToHost);

	/* Free Device Memory*/
	hipFree(d_A);
	hipFree(d_X);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float timeElapsed;
	hipEventElapsedTime (&timeElapsed, start, stop);
	printf("Time Elapsed : %f ms\n", timeElapsed);
}



int main() {
	char UPLO, TRANS, DIAG; 
	int N; int Incx;
	double * A, *X;
	/* 
		Test Cases are to be input
	*/
	
	scanf("%c %c %c %d",&UPLO,&TRANS, &DIAG,  &N);
	
	int size_a = (N* (N+1))/2;
	hipHostMalloc((void**) &A, sizeof(double)*size_a , hipHostMallocDefault);
	hipHostMalloc((void**) &X, sizeof(double)*N, hipHostMallocDefault);
	
	for (int i =0; i<N;i++) {
		for(int j = 0;j<N;j++) {
			if(i>j) {continue;}
			scanf("%lf", &A[i + (j*(j+1))/2]);
		}
	}
	
	for (int i= 0; i<N;i++) {
		scanf("%lf", X + i);
	}

	dtpmv(UPLO, TRANS, DIAG, N, A, X);

	// Display Outpu
	FILE * fp;
	fp = fopen("Results/dtpmv_cublas_last.txt", "w");
	for(int i =0; i<N;i++) {
		fprintf(fp,"%f ", X[i]);
	}
	fclose(fp);

}
