#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include <hip/hip_runtime_api.h>
#define BLOCK_SIZE 16 // Threads per block supported by the GPU


__global__ void dtpmv_kernel ( char UPLO, char TRANS, char DIAG,int N,double * A, double *X , double *T) {
	int elementId = blockIdx.x * BLOCK_SIZE + threadIdx.x; 

	if(elementId>=N) {
		return;
	}
	double sum = 0;
	if (UPLO == 'U' && TRANS == 'N' ) {
		for (int i = elementId; i<N ; i++) {
			sum += A[elementId *N + i - ((elementId+1)*elementId)/2 ] * X[i];
		}
	} else if (UPLO == 'L' && TRANS == 'N') {

		int temp = (elementId * (elementId +1)) /2;
		for (int i = 0; i <=elementId; i++ ) {
			sum += A[temp+i] * X[i];
		}

	} else if (UPLO == 'U' && TRANS == 'T') {
		for (int i = elementId;i>=0; i--) {
			sum += A[i*N + elementId- ((i+1)*i)/2]*X[i];
		}
	} else {
		for (int i = elementId;i < N; i++) {
			sum += A[i*N + elementId- ((2*N-i-1)*i)/2 ]*X[i];;
		}
	}
	
	T[elementId] = sum;
}

int Ceil (int n , int m) {
	if (n%m == 0) {
		return n/m;
	} else {
		return n/m +1;
	}
}

void dtpmv(char UPLO, char TRANS, char DIAG, int N, double * A, double * X) {

	double * d_A, *d_X, *d_T;
	int size_a = (N*(N+1))/2;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	hipMalloc((void**) &d_A, sizeof(double)*size_a );
	hipMalloc((void**) &d_X, sizeof(double)*N );
	hipMalloc((void**) &d_T, sizeof(double)*N );

	hipMemcpy(d_A, A , sizeof(double) *size_a, hipMemcpyHostToDevice);
	hipMemcpy(d_X, X, sizeof(double) * N, hipMemcpyHostToDevice);
	// cudaMemcpy(d_T, X, sizeof(double) * N, cudaMemcpyHostToDevice);
	
	/*Determine Kernel Parameters*/
	int num_blocks = Ceil(N, BLOCK_SIZE);

	/*Launch Kernel*/
	dtpmv_kernel <<<num_blocks, BLOCK_SIZE >>> (UPLO, TRANS, DIAG, N, d_A, d_X, d_T);


	/* Copy Memory Back to Host */
	hipMemcpy(X, d_T, sizeof(double) * N , hipMemcpyDeviceToHost);


	/* Free Device Memory*/
	hipFree(d_A);
	hipFree(d_X);
	hipFree(d_T);

	hipEventRecord(stop, 0);
	float timeElapsed;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timeElapsed, start, stop);
	printf("Time Elapsed : %f\n", timeElapsed);
}



int main() {
	char UPLO, TRANS, DIAG; 
	int N; int Incx;
	double * A, *X;
	/* 
		Test Cases are to be input
	*/
	
	scanf("%c %c %c %d",&UPLO,&TRANS, &DIAG,  &N);
	
	int size_a = (N* (N+1))/2;
	hipHostMalloc((void**) &A, sizeof(double)*size_a , hipHostMallocDefault);
	hipHostMalloc((void**) &X, sizeof(double)*N, hipHostMallocDefault);
	
	for (int i= 0; i<size_a;i++) {
		scanf("%lf", &A[i]);
	}

	for (int i= 0; i<N;i++) {
		scanf("%lf", X + i);
	}

	dtpmv(UPLO, TRANS, DIAG, N, A, X);

	/* Display Output */
	FILE * fp;
	fp = fopen("Results/dtpmv_gpu_last.txt", "w");
	for(int i =0; i<N;i++) {
		fprintf(fp,"%f ", X[i]);
	}
	fclose(fp);
}
